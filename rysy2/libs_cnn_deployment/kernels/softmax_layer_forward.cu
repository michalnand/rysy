#include "hip/hip_runtime.h"
#include "convolution_layer_forward.cuh"

#include "../cuda_float_allocator.cuh"

__global__
void cuda_softmax_forward_kernel(  float *output,
                                   float *input,
                                   sShape input_shape)
{
    unsigned int x      = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int y      = threadIdx.y + blockIdx.y*blockDim.y;

    if (y < input_shape.h)
    if (x < input_shape.w)
    {
        unsigned int input_size = input_shape.w*input_shape.h;
        unsigned int input_idx;

        float max = -10000000.0;

        input_idx  = y*input_shape.w + x;
        for (unsigned int filter = 0; filter < input_shape.d; filter++)
        {
            if (input[input_idx] > max)
                max = input[input_idx];

            input_idx+= input_size;
        }

        input_idx  = y*input_shape.w + x;
        float sum = 0.00000001;
        for (unsigned int filter = 0; filter < input_shape.d; filter++)
        {
            sum+= exp(input[input_idx] - max);
            input_idx+= input_size;
        }

        input_idx  = y*input_shape.w + x;
        for (unsigned int filter = 0; filter < input_shape.d; filter++)
        {
            output[input_idx] = exp(input[input_idx] - max)/sum;
            input_idx+= input_size;
        }
    }
}


void softmax_layer_forward( float *output, float *input,
                            sShape input_shape)
{
    unsigned int input_size_y = input_shape.h;
    unsigned int input_size_x = input_shape.w;

    dim3 block(8, 8);
    dim3 grid((input_size_x  + block.x + 1)/block.x,
              (input_size_y  + block.y + 1)/block.y);

    cuda_softmax_forward_kernel<<<grid, block>>>(   output,
                                                    input,
                                                    input_shape);
    hipDeviceSynchronize();
}
