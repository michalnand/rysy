#include "hip/hip_runtime.h"
#include "activation_relu_layer.cuh"

__host__
void cpu_activation_relu_forward_kernel(float *output, float *input, unsigned int size)
{
    for (unsigned int idx = 0; idx < size; idx++)
    {
        float tmp = input[idx];

        if (tmp < 0.0)
            tmp = 0.0;

        output[idx] = tmp;
    }
}

__global__
void cuda_activation_relu_forward_kernel(float *output, float *input, unsigned int size)
{
    unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;

    if (idx < size)
    {
        float tmp = input[idx];

        if (tmp < 0.0)
            tmp = 0.0;

        output[idx] = tmp;
    }
}

void activation_relu_layer_forward(  Tensor &output, Tensor &input)
{
    unsigned int size = output.size();

    #ifdef NETWORK_USE_CUDA

        dim3 block(16);
        dim3 grid((size + block.x + 1)/block.x);

        cuda_activation_relu_forward_kernel<<<grid, block>>>(output.v, input.v, size);
        hipDeviceSynchronize();

    #else

        cpu_activation_relu_forward_kernel(output.v, input.v, size);

    #endif
}


__host__
void cpu_activation_relu_backward_kernel(float *error_back, float *error, float *output, unsigned int size)
{
  for (unsigned int idx = 0; idx < size; idx++)
  {
    if (output[idx] > 0.0)
      error_back[idx] = error[idx];
    else
      error_back[idx] = 0.0;
  }
}

__global__
void cuda_activation_relu_backward_kernel(float *error_back, float *error, float *output, unsigned int size)
{
    unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;

    if (idx < size)
    {
        if (output[idx] > 0.0)
            error_back[idx] = error[idx];
        else
            error_back[idx] = 0.0;
    }
}

void activation_relu_layer_backward( Tensor &error_back, Tensor &output, Tensor &error)
{
    unsigned int size = output.size();

    #ifdef NETWORK_USE_CUDA

        dim3 block(16);
        dim3 grid((size + block.x + 1)/block.x);

        cuda_activation_relu_backward_kernel<<<grid, block>>>(error_back.v, error.v, output.v, size);
        hipDeviceSynchronize();
    #else

        cpu_activation_relu_backward_kernel(error_back.v, error.v, output.v, size);

    #endif
}
