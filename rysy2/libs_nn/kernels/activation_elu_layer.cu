#include "hip/hip_runtime.h"
#include "activation_elu_layer.cuh"

#define ELU_ALPHA ((float)1.0)

__host__
void cpu_activation_elu_forward_kernel(float *output, float *input, unsigned int size)
{
    for (unsigned int idx = 0; idx < size; idx++)
    {
        float tmp = input[idx];

        if (tmp <= 0.0)
            tmp = ELU_ALPHA*(exp(tmp) - 1.0);

        output[idx] = tmp;
    }
}

__global__
void cuda_activation_elu_forward_kernel(float *output, float *input, unsigned int size)
{
    unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;

    if (idx < size)
    {
        float tmp = input[idx];

        if (tmp <= 0.0)
            tmp = ELU_ALPHA*(exp(tmp) - 1.0);

        output[idx] = tmp;
    }
}

void activation_elu_layer_forward(  Tensor &output, Tensor &input)
{
    unsigned int size = output.size();

    #ifdef NETWORK_USE_CUDA

        dim3 block(16);
        dim3 grid((size + block.x + 1)/block.x);

        cuda_activation_elu_forward_kernel<<<grid, block>>>(output.v, input.v, size);
        hipDeviceSynchronize();

    #else

        cpu_activation_elu_forward_kernel(output.v, input.v, size);

    #endif
}


__host__
void cpu_activation_elu_backward_kernel(float *error_back, float *error, float *output, unsigned int size)
{
    for (unsigned int idx = 0; idx < size; idx++)
    {
        if (output[idx] > 0.0)
            error_back[idx] = error[idx];
        else
            error_back[idx] = (output[idx] + ELU_ALPHA)*error[idx];
    }
}

__global__
void cuda_activation_elu_backward_kernel(float *error_back, float *error, float *output, unsigned int size)
{
    unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;

    if (idx < size)
    {
        if (output[idx] > 0.0)
            error_back[idx] = error[idx];
        else
            error_back[idx] = (output[idx] + ELU_ALPHA)*error[idx];
    }
}

void activation_elu_layer_backward( Tensor &error_back, Tensor &output, Tensor &error)
{
    unsigned int size = output.size();

    #ifdef NETWORK_USE_CUDA

        dim3 block(16);
        dim3 grid((size + block.x + 1)/block.x);

        cuda_activation_elu_backward_kernel<<<grid, block>>>(error_back.v, error.v, output.v, size);
        hipDeviceSynchronize();

    #else

        cpu_activation_elu_backward_kernel(error_back.v, error.v, output.v, size);

    #endif
}
