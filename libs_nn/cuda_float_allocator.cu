#include "cuda_float_allocator.cuh"
#include <iostream>

/*
CudaFloatAllocator cuda_float_allocator;


CudaFloatAllocator::CudaFloatAllocator()
{

}

CudaFloatAllocator::~CudaFloatAllocator()
{

}

float* CudaFloatAllocator::malloc(unsigned int count)
{
  mutex.lock();

  float *result = nullptr;

  hipMalloc(&result, count*sizeof(float));

  mutex.unlock();

  clear(result, count);

  return result;
}

void CudaFloatAllocator::free(void *ptr)
{
  mutex.lock();

  if (ptr != nullptr)
  {
    hipFree(ptr);
    ptr = nullptr;
  }

  mutex.unlock();
}

void CudaFloatAllocator::host_to_device(float *dev_ptr, float *host_ptr, unsigned int size)
{
  hipMemcpy(dev_ptr, host_ptr, size*sizeof(float), hipMemcpyHostToDevice);
}

void CudaFloatAllocator::device_to_host(float *host_ptr, float *dev_ptr, unsigned int size)
{
  hipMemcpy(host_ptr, dev_ptr, size*sizeof(float), hipMemcpyDeviceToHost);
}

void CudaFloatAllocator::device_to_device(float *dest_ptr, float *src_ptr, unsigned int size)
{
  hipMemcpy(dest_ptr, src_ptr, size*sizeof(float), hipMemcpyDeviceToDevice);
}

void CudaFloatAllocator::clear(float *result, unsigned int size)
{
  hipMemset(result, 0, size*sizeof(float));
}
*/







float* cu_malloc(unsigned int count)
{
  float *result = nullptr;

  hipMalloc(&result, count*sizeof(float));


  cu_clear(result, count);

  return result;
}

void cu_free(void *ptr)
{
  if (ptr != nullptr)
  {
    hipFree(ptr);
    ptr = nullptr;
  }
}

void cu_host_to_device(float *dev_ptr, float *host_ptr, unsigned int size)
{
  hipMemcpy(dev_ptr, host_ptr, size*sizeof(float), hipMemcpyHostToDevice);
}

void cu_device_to_host(float *host_ptr, float *dev_ptr, unsigned int size)
{
  hipMemcpy(host_ptr, dev_ptr, size*sizeof(float), hipMemcpyDeviceToHost);
}

void cu_device_to_device(float *dest_ptr, float *src_ptr, unsigned int size)
{
  hipMemcpy(dest_ptr, src_ptr, size*sizeof(float), hipMemcpyDeviceToDevice);
}

void cu_clear(float *result, unsigned int size)
{
  hipMemset(result, 0, size*sizeof(float));
}
