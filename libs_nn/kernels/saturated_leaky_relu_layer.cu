#include "hip/hip_runtime.h"
#include "saturated_leaky_relu_layer.cuh"


#define RELU_LEAK           ((float)1.0/16.0)
#define RELU_LIMIT_VALUE    ((float)1.0)

__host__
void cpu_saturated_leaky_relu_layer_forward_kernel(float *output, float *input, unsigned int size)
{
  for (unsigned int idx = 0; idx < size; idx++)
  {
    float tmp = input[idx];

    if (tmp < 0.0)
        tmp = RELU_LEAK*RELU_LEAK;

    if (tmp > RELU_LIMIT_VALUE)
        tmp = RELU_LIMIT_VALUE;

    if (tmp < -RELU_LIMIT_VALUE)
        tmp = -RELU_LIMIT_VALUE;

    output[idx] = tmp;
  }
}

__global__
void cuda_saturated_leaky_relu_layer_forward_kernel(float *output, float *input, unsigned int size)
{
  unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;

  if (idx < size)
  {
    float tmp = input[idx];

    if (tmp < 0.0)
        tmp = RELU_LEAK*RELU_LEAK;

    if (tmp > RELU_LIMIT_VALUE)
        tmp = RELU_LIMIT_VALUE;

    if (tmp < -RELU_LIMIT_VALUE)
        tmp = -RELU_LIMIT_VALUE;

    output[idx] = tmp;
  }
}

void saturated_leaky_relu_layer_forward(  Tensor &output, Tensor &input)
{
  unsigned int size = output.size();

  #ifdef NETWORK_USE_CUDA

    unsigned int block_size = 16;
    if (size > 256)
      block_size = 256;

    dim3 block(block_size);
    dim3 grid((size + block.x - 1)/block.x);

    cuda_saturated_leaky_relu_layer_forward_kernel<<<grid, block>>>(output.v, input.v, size);
    hipDeviceSynchronize();

  #else

    cpu_saturated_leaky_relu_layer_forward_kernel(output.v, input.v, size);

  #endif
}


__host__
void cpu_saturated_leaky_relu_layer_backward_kernel(float *error_back, float *error, float *output, unsigned int size)
{
    for (unsigned int idx = 0; idx < size; idx++)
    {
        if (output[idx] > 0.0)
            error_back[idx] = error[idx];
        else
            error_back[idx] = RELU_LEAK*error[idx];
    }
}

__global__
void cuda_saturated_leaky_relu_layer_backward_kernel(float *error_back, float *error, float *output, unsigned int size)
{
    unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;

    if (idx < size)
    {
        if (output[idx] > 0.0)
            error_back[idx] = error[idx];
        else
            error_back[idx] = RELU_LEAK*error[idx];
    }
}

void saturated_leaky_relu_layer_backward( Tensor &error_back, Tensor &output, Tensor &error)
{
  unsigned int size = output.size();

  #ifdef NETWORK_USE_CUDA

      unsigned int block_size = 16;
      if (size >= 256)
        block_size = 256;

      dim3 block(block_size);
      dim3 grid((size + block.x - 1)/block.x);

      cuda_saturated_leaky_relu_layer_backward_kernel<<<grid, block>>>(error_back.v, error.v, output.v, size);
      hipDeviceSynchronize();
  #else

    cpu_saturated_leaky_relu_layer_backward_kernel(error_back.v, error.v, output.v, size);

  #endif
}
