#include "hip/hip_runtime.h"
#include "relu6_layer.cuh"


#define RELU_LIMIT_VALUE    ((float)6.0)

__host__
void cpu_relu_6_forward_kernel(float *output, float *input, unsigned int size)
{
  for (unsigned int idx = 0; idx < size; idx++)
  {
    float tmp = input[idx];

    if (tmp < 0.0)
        tmp = 0.0;

    if (tmp > RELU_LIMIT_VALUE)
        tmp = RELU_LIMIT_VALUE;

    output[idx] = tmp;
  }
}

__global__
void cuda_relu_6_forward_kernel(float *output, float *input, unsigned int size)
{
  unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;

  if (idx < size)
  {
    float tmp = input[idx];

    if (tmp < 0.0)
        tmp = 0.0;

    if (tmp > RELU_LIMIT_VALUE)
          tmp = RELU_LIMIT_VALUE;

    output[idx] = tmp;
  }
}

void relu_6_layer_forward(  Tensor &output, Tensor &input)
{
  unsigned int size = output.size();

  #ifdef NETWORK_USE_CUDA

    unsigned int block_size = 16;
    if (size > 256)
      block_size = 256;

    dim3 block(block_size);
    dim3 grid((size + block.x - 1)/block.x);

    cuda_relu_6_forward_kernel<<<grid, block>>>(output.v, input.v, size);
    hipDeviceSynchronize();

  #else

    cpu_relu_6_forward_kernel(output.v, input.v, size);

  #endif
}


__host__
void cpu_relu_6_backward_kernel(float *error_back, float *error, float *output, unsigned int size)
{
  for (unsigned int idx = 0; idx < size; idx++)
  {
    if ((output[idx] > 0.0)&&(output[idx] < RELU_LIMIT_VALUE))
      error_back[idx] = error[idx];
    else
      error_back[idx] = 0.0;
  }
}

__global__
void cuda_relu_6_backward_kernel(float *error_back, float *error, float *output, unsigned int size)
{
  unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;

  if (idx < size)
  {
    if ((output[idx] > 0.0)&&(output[idx] < RELU_LIMIT_VALUE))
      error_back[idx] = error[idx];
    else
      error_back[idx] = 0.0;
  }
}

void relu_6_layer_backward( Tensor &error_back, Tensor &output, Tensor &error)
{
  unsigned int size = output.size();

  #ifdef NETWORK_USE_CUDA

      unsigned int block_size = 16;
      if (size >= 256)
        block_size = 256;

      dim3 block(block_size);
      dim3 grid((size + block.x - 1)/block.x);

      cuda_relu_6_backward_kernel<<<grid, block>>>(error_back.v, error.v, output.v, size);
      hipDeviceSynchronize();
  #else

    cpu_relu_6_backward_kernel(error_back.v, error.v, output.v, size);

  #endif
}
