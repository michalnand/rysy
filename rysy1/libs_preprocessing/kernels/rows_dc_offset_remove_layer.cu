#include "hip/hip_runtime.h"
#include "rows_dc_offset_remove_layer.cuh"



__host__
void cpu_rows_dc_offset_remove_layer_kernel( float *output, float *input,
                                        unsigned int width, unsigned height, unsigned int depth)
{
  for (unsigned int channel = 0; channel < depth; channel++)
  for (unsigned int row = 0; row < height; row++)
  for (unsigned int column = 0; column < (width-1); column++)
  {
    unsigned int idx = (channel*height + row)*width + column;

    output[idx] = input[idx] - input[idx+1];
  }
}

__global__
void cuda_rows_dc_offset_remove_layer_kernel( float *output, float *input,
                                        unsigned int width, unsigned int height, unsigned int depth)
{
  unsigned int column     = threadIdx.x + blockIdx.x*blockDim.x;
  unsigned int row        = threadIdx.y + blockIdx.y*blockDim.y;
  unsigned int channel    = threadIdx.z + blockIdx.z*blockDim.z;

  if (channel < depth)
  if (row < height)
  if (column < (width-1))
  {
    unsigned int idx = (channel*height + row)*width + column;

    output[idx] = input[idx] - input[idx+1];
  }
}



void rows_dc_offset_remove_layer(Tensor &output, Tensor &input)
{
  output.clear();
  
  #ifdef NETWORK_USE_CUDA

    dim3 block(8, 8, 1);
    dim3 grid((input.w() + block.x - 1)/block.x,
              (input.h() + block.y - 1)/block.y,
              (input.d() + block.z - 1)/block.z);

    cuda_rows_dc_offset_remove_layer_kernel<<<grid, block>>>(output.v, input.v, input.w(), input.h(), input.d());
    hipDeviceSynchronize();
  #else

    cpu_rows_dc_offset_remove_layer_kernel(output.v, input.v, input.w(), input.h(), input.d());

  #endif
}
